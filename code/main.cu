#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include"logger.h"
#include"parser_dense.h"
#include"parser.h"
#include"data.h"
#include <fstream>
#include"kernelgraph.h"
#include"config.h"
#include <chrono>
using namespace std;
using std::chrono::high_resolution_clock;
using std::chrono::milliseconds;

std::unique_ptr<Data> data;
std::unique_ptr<GraphWrapper> graph; 
int topk = 0;
int dim_size = 0;
int display_topk = 1;

void build_callback(idx_t idx,std::vector<std::pair<int,value_t>> point){
    data->add(idx,point);
    // graph->add_vertex(idx,point);
}

std::vector<std::vector<std::pair<int,value_t>>> batch_queries;
std::vector<std::vector<idx_t>> results(ACC_BATCH_SIZE);

void flush_queries(){
	results.resize(batch_queries.size());
	const int repeat = 1; // NOTICE: You can repeat multiple times to have an average search performance
	for(int i = 0;i < repeat;++i)
	    graph->search_top_k_batch(batch_queries,topk,results);
    // for(int i = 0;i < 1;++i){
    // // for(int i = 0;i < batch_queries.size();++i){
    //     auto& result = results[i];
    //     for(int i = 0;i < result.size() && i < display_topk;++i)
    //         printf("%zu ",result[i]);
    //     printf("\n");
    // }
    std::ofstream out("res_cifar10_s2_200_" + std::to_string(display_topk) + ".txt");
    for (int i = 0; i < batch_queries.size(); i++) {
        int s = 0;
        for (int j = 0; j < batch_queries[i].size(); j++) {
            for (int k = s; k < batch_queries[i][j].first; k++) {
                out<< 0 <<"_";
            }
            out<< batch_queries[i][j].second <<"_";
            s = batch_queries[i][j].first + 1;
        }
        for (int k = s; k < dim_size; k++) {
            out<< 0 <<"_";
        }
        out<<'\n';
        for (int j = 1; j < 101; j++) {
            out<< results[i][j] <<" ";
        }
        out<<'\n';
    }
    batch_queries.clear();
}

void query_callback(idx_t idx,std::vector<std::pair<int,value_t>> point){
    batch_queries.push_back(point);
	// Uncomment the following lines to have a finer granularity batch processing
    //if(batch_queries.size() == ACC_BATCH_SIZE){
    //    flush_queries();
    //}
	/////////////////////
}




void usage(char** argv){
    printf("Usage: %s <build/test> <build_data> <query_data> <search_top_k> <row> <dim> <return_top_k> <l2/ip/cos>\n",argv[0]);
}

int main(int argc,char** argv){
    if(argc != 9){
        usage(argv);
        return 1;
    }
	// You may need to increase this parameter for some new GPUs
    hipDeviceSetLimit(hipLimitMallocHeapSize,800*1024*1024);
	//////////////////////
	size_t row = atoll(argv[5]);
	int dim = atoi(argv[6]);
    dim_size = dim;
	display_topk = atoi(argv[7]);
	std::string dist_type = argv[8];
	data = std::unique_ptr<Data>(new Data(row,dim));
	if(dist_type == "l2"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<0>(data.get())); 
	}else if(dist_type == "ip"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<1>(data.get())); 
	}else if(dist_type == "cos"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<2>(data.get())); 
	}else{
		usage(argv);
		return 1;
	}
    std::string mode = std::string(argv[1]);
    topk = atoi(argv[4]);
    if(mode == "build"){
        high_resolution_clock::time_point beginTime = high_resolution_clock::now();
        //std::unique_ptr<ParserDense> build_parser(new ParserDense(argv[2],build_callback));
        std::unique_ptr<Parser> build_parser(new Parser(argv[2],build_callback));
        fprintf(stderr,"Writing the graph and data...");    
        high_resolution_clock::time_point beginTime1 = high_resolution_clock::now();
        // For s2, you can use below codes to run under the s2 setting.
        graph->init_build_v2();
        high_resolution_clock::time_point endTime1 = high_resolution_clock::now();
        cout << "Running Time:" << std::chrono::duration_cast<milliseconds>(endTime1 - beginTime1).count()  << "ms" << endl;
        graph->fullybuild_v2();

        // For s1, you can use below codes to run under the s1 setting.

        // graph->init_build_v2();
        // high_resolution_clock::time_point endTime1 = high_resolution_clock::now();
        // cout << "Running Time:" << std::chrono::duration_cast<milliseconds>(endTime1 - beginTime1).count()  << "ms" << endl;
        // graph->fullybuild_v2();
        
        data->dump();
        fprintf(stderr,"...");    
        graph->dump();
        fprintf(stderr,"done\n");
        high_resolution_clock::time_point endTime = high_resolution_clock::now();
        milliseconds timeInterval = std::chrono::duration_cast<milliseconds>(endTime - beginTime);
        cout << "Running Time:" << timeInterval.count()  << "ms" << endl;
    }else if(mode == "test"){
        fprintf(stderr,"Loading the graph and data...");    
        data->load();
        fprintf(stderr,"...");    
        graph->load();
        fprintf(stderr,"done\n");    
        //std::unique_ptr<ParserDense> query_parser(new ParserDense(argv[3],query_callback));
        std::unique_ptr<Parser> query_parser(new Parser(argv[3],query_callback));
		flush_queries();	
    }else{
        usage(argv);
        return 1;
    }
    return 0;
}
